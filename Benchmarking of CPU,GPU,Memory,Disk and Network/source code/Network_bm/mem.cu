#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<string.h>
#include<hip/hip_runtime.h>

__global__ void func(int *a,int *b,int *c)
{
int t=blockIdx.x*blockDim.x+threadIdx.x;
c[t]=a[t]+b[t];
}

int main( void )
{

int mem_size=10000000;
long blocksize=1;
double t;
long s,e;
char *c;
int i=0;
struct timeval start,end;
char *dev_a;
c=(char *)malloc(blocksize*sizeof(char));
memset(c,'#',blocksize);
hipMalloc( (void**)&dev_a, blocksize * sizeof(int) );
//cudaMalloc( (void**)&dev_b, n * sizeof(int) );

gettimeofday(&start,NULL);
s=start.tv_sec*1000000 + start.tv_usec;//get start time
for(i=0;i<(mem_size/blocksize);i++)
{
hipMemcpy( dev_a, c, blocksize * sizeof(char),hipMemcpyHostToDevice );
}
//cudaMemcpy( dev_b, b, n * sizeof(int),cudaMemcpyHostToDevice );

gettimeofday(&end,NULL);
e=end.tv_sec*1000000 + end.tv_usec;//get end time
 // copy the array 'c' back from the GPU to the CPU
t=e-s;
printf("Time taken: %lf",t);
double d=(double)mem_size/1000.0;
d=d/(double)t;
printf("throughput: %lf gb/s",d);

 hipFree( dev_a );
 
 return(0);
}

