#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<string.h>
#include<hip/hip_runtime.h>

__global__ void func(int *a,int *b,int *c)
{
int t=blockIdx.x*blockDim.x+threadIdx.x;
c[t]=a[t]+b[t]+1;
c[t]=a[t]+b[t];
c[t]=a[t]+b[t];
c[t]=a[t]+b[t];
c[t]=a[t]+b[t];
}

int main( void )
{
int cyclecount=100000;
int devicecount,device;
double time_s;
int blocks,threads,n;
long start_time,end_time;
struct hipDeviceProp_t properties;
int *a, *b, *c;
struct timeval start,stop;
int *dev_a, *dev_b, *dev_c;
hipError_t cudaResultCode = hipGetDeviceCount(&devicecount);
if (cudaResultCode != hipSuccess)
    devicecount = 0;
/* machines with no GPUs can still report one emulation device */
for (device = 0; device < devicecount; ++device) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) /* 9999 means emulation only */
    if (device==0)
    {
            printf("multiProcessorCount %d\n",properties.multiProcessorCount);
            printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
            blocks=properties.multiProcessorCount;
            threads=properties.maxThreadsPerMultiProcessor;
            n=properties.multiProcessorCount * properties.maxThreadsPerMultiProcessor;
    }
}
//printf()
a=(int*)malloc(n * sizeof(int));
b=(int*)malloc(n * sizeof(int));
c=(int*)malloc(n * sizeof(int));

 // allocate the memory on the GPU
hipMalloc( (void**)&dev_a, n * sizeof(int) );
hipMalloc( (void**)&dev_b, n * sizeof(int) );
hipMalloc( (void**)&dev_c, n * sizeof(int) );
 // fill the arrays 'a' and 'b' on the CPU
 for (int i=0; i<n; i++) {
 a[i] = -i;
 b[i] = i * i;
 }
hipMemcpy( dev_a, a, n * sizeof(int),hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, n * sizeof(int),hipMemcpyHostToDevice );
gettimeofday(&start,NULL);
int l;
start_time=start.tv_sec*1000000 + start.tv_usec;//get start time
for(l=0;l<cyclecount;l++)
 func<<<blocks,threads>>>( dev_a, dev_b, dev_c );
gettimeofday(&stop,NULL);
end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time
 // copy the array 'c' back from the GPU to the CPU
hipMemcpy( c, dev_c, n * sizeof(int),hipMemcpyDeviceToHost );
 // display the results
// for (int i=0; i<N; i++) {
// printf( "%d + %d = %d\n", a[i], b[i], c[i] );
// }
 // free the memory allocated on the GPU
time_s=end_time-start_time;
printf("GFLOPS: %lf",(double)(cyclecount*n*5)/((double)time_s*1000.0));

 hipFree( dev_a );
 hipFree( dev_b );
 hipFree( dev_c );
 return(0);
}

