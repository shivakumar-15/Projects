#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<string.h>
#include<hip/hip_runtime.h>

__global__ void funcflops(float *a,float *b,float *c)
{
int t=blockIdx.x*blockDim.x+threadIdx.x;
c[t]=a[t]+b[t]+1;
c[t+2]=a[t+2]*b[t+2];
c[t]=a[t]+b[t];
c[t]=a[t]+b[t];
c[t]=a[t]+b[t];
//c[t]=a[t]*b[t];
//c[t]=a[t]-b[t];
//c[t]=a[t]/b[t];
}

int main( void )
{

int cyclecount=10000;
int devicecount,device;
double time_s;
int blocks,threads,n;
long start_time,end_time;
struct hipDeviceProp_t properties;
float *a, *b, *c;
struct timeval start,stop;
float *dev_a, *dev_b, *dev_c;
hipError_t cudaResultCode = hipGetDeviceCount(&devicecount);
if (cudaResultCode != hipSuccess)
    devicecount = 0;
/* machines with no GPUs can still report one emulation device */
for (device = 0; device < devicecount; ++device) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) /* 9999 means emulation only */
    if (device==0)
    {
            printf("multiProcessorCount %d\n",properties.multiProcessorCount);
            printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
            blocks=properties.multiProcessorCount;
            threads=properties.maxThreadsPerMultiProcessor;
//            n=properties.multiProcessorCount * properties.maxThreadsPerMultiProcessor;
	    n=blocks*threads;  
  }
}
printf("%s\n",properties.name);
a=(float*)malloc(n * sizeof(float));
b=(float*)malloc(n * sizeof(float));
c=(float*)malloc(n * sizeof(float));

 // allocate the memory on the GPU
hipMalloc( (void**)&dev_a, n * sizeof(float) );
hipMalloc( (void**)&dev_b, n * sizeof(float) );
hipMalloc( (void**)&dev_c, n * sizeof(float) );
 // fill the arrays 'a' and 'b' on the CPU
 for (int i=0; i<n; i++) {
 a[i] = -i;
 b[i] = i * i;
 }
hipMemcpy( dev_a, a, n * sizeof(int),hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, n * sizeof(int),hipMemcpyHostToDevice );
gettimeofday(&start,NULL);
int i;
start_time=start.tv_sec*1000000 + start.tv_usec;//get start time
for(i=0;i<cyclecount;i++)
 funcflops<<<blocks,threads>>>( dev_a, dev_b, dev_c );
gettimeofday(&stop,NULL);
end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time
 // copy the array 'c' back from the GPU to the CPU
hipMemcpy( c, dev_c, n * sizeof(int),hipMemcpyDeviceToHost );
 // display the results
// for (int i=0; i<N; i++) {
// printf( "%d + %d = %d\n", a[i], b[i], c[i] );
// }
 // free the memory allocated on the GPU
time_s=end_time-start_time;
printf("Time taken: %lf\n",time_s);
//printf("GFLOPS: %lf\n",);
double d=(double)(cyclecount*n*5)/((double)time_s*1000.0);
//d=d/1000000.0;
printf("GFlops  %lf \n",d);
 hipFree( dev_a );
 hipFree( dev_b );
 hipFree( dev_c );
 return(0);
}

