#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<string.h>
#include<hip/hip_runtime.h>

__global__ void funcflops(float *a,float *b,float *c)
{
int t=blockIdx.x*blockDim.x+threadIdx.x;
c[t]=a[t]+b[t];
}

int main( void )
{
int cyclecount=100000;
int devicecount,device;
double time_s;
int blocks,threads,n;
long start_time,end_time;
struct hipDeviceProp_t properties;
float *a, *b, *c;
struct timeval start,stop;
float *dev_a, *dev_b, *dev_c;
hipError_t cudaResultCode = hipGetDeviceCount(&devicecount);
if (cudaResultCode != hipSuccess)
    devicecount = 0;
/* machines with no GPUs can still report one emulation device */
for (device = 0; device < devicecount; ++device) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) /* 9999 means emulation only */
    if (device==0)
    {
            printf("multiProcessorCount %d\n",properties.multiProcessorCount);
            printf("maxThreadsPerMultiProcessor %d\n",properties.maxThreadsPerMultiProcessor);
            blocks=properties.multiProcessorCount;
            threads=properties.maxThreadsPerMultiProcessor;
            n=properties.multiProcessorCount * properties.maxThreadsPerMultiProcessor;
    }
}
printf("%s\n",properties.name);
a=(float*)malloc(n * sizeof(float));
b=(float*)malloc(n * sizeof(float));
c=(float*)malloc(n * sizeof(float));

 // allocate the memory on the GPU
hipMalloc( (void**)&dev_a, n * sizeof(float) );
hipMalloc( (void**)&dev_b, n * sizeof(float) );
hipMalloc( (void**)&dev_c, n * sizeof(float) );
 // fill the arrays 'a' and 'b' on the CPU
 for (int i=0; i<n; i++) {
 a[i] = -i;
 b[i] = i * i;
 }
hipMemcpy( dev_a, a, n * sizeof(int),hipMemcpyHostToDevice );
hipMemcpy( dev_b, b, n * sizeof(int),hipMemcpyHostToDevice );
gettimeofday(&start,NULL);
int l;
start_time=start.tv_sec*1000000 + start.tv_usec;//get start time
for(l=0;l<cyclecount;l++)
 funcflops<<<blocks,threads>>>( dev_a, dev_b, dev_c );
gettimeofday(&stop,NULL);
end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time
 // copy the array 'c' back from the GPU to the CPU
hipMemcpy( c, dev_c, n * sizeof(int),hipMemcpyDeviceToHost );
 // display the results
// for (int i=0; i<N; i++) {
// printf( "%d + %d = %d\n", a[i], b[i], c[i] );
// }
 // free the memory allocated on the GPU
time_s=end_time-start_time;
printf("Time taken: %lf",time_s);
printf("GFLOPS: %lf",(double)(cyclecount*n*3)/(time_s*1000000000));

 hipFree( dev_a );
 hipFree( dev_b );
 hipFree( dev_c );
 return(0);
}

